#include "hip/hip_runtime.h"
/*
* Volume of a D-dimensional Hypersphere
* see: https://cs.binghamton.edu/~kchiu/cs447/assign/final/
* 
* Graduate student @ Thomas J. Watson College of Engineering and Applied
* Sciences, Binghamton University.
*
* author: Gregory Maldonado
* email : gmaldonado@cs.binghamton.edu
* date  : 2024-05-05
* web   : https://cs.binghamton.edu/~gmaldonado/
*`
* file name:   chypersphere.cu
* description: D-dimensional Hypersphere computation using CUDA
*
*  THE SOFTWARE IS PROVIDED “AS IS”, WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
* IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
* FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
* AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
* LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
* OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
* SOFTWARE.
*/

#include<stdio.h>
#include<stdlib.h>
#include <iostream>
#include <vector>
#include <random>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include "chypersphere.h"

//==================================================================== 80 ====>>

__global__ void compute_distances(int max_samples, 
                                    int max_dimensions,
                                    double* random_numbers,
                                    double* distances) {

   for (int i = 0; i < 1; ++i) {
      for (int j = 0; j < max_dimensions; ++j) {
         printf("%f => %f\n", random_numbers[(max_dimensions * i) + j], random_numbers[(max_dimensions * i) + j] * random_numbers[(max_dimensions * i) + j]);
         distances[i] += (random_numbers[(max_dimensions * i) + j] * random_numbers[(max_dimensions * i) + j]);
      }
   } 
}

void compute(const uint8_t min_dimensions, const uint8_t max_dimensions,
               const size_t max_samples) {
   
   // generate random numbers using std::random_device
   // https://stackoverflow.com/questions/1340729
   std::random_device rd;
   std::mt19937 e2(rd());
   std::uniform_real_distribution<double> distribution(0.0, 1.0);

   double* random_numbers;
   double* random_numbers_d;
   
   double * distances;
   double * distances_d;

   // https://stackoverflow.com/questions/9373929/cuda-transfer-2d-array-from-host-to-device
   random_numbers = (double*) malloc( max_dimensions * max_samples * sizeof(double));
   hipMalloc(&random_numbers_d, max_dimensions * max_samples * sizeof(double));
   
   distances = (double*) malloc(max_samples * sizeof(double));
   hipMalloc(&distances_d, max_samples * sizeof(double));
   
   for (size_t sample = 0; sample < max_samples; ++sample) {
      for (size_t dim = 0; dim < max_dimensions; ++dim) {
         random_numbers[(max_dimensions * sample) + dim] = distribution(e2);
         std::cout << random_numbers[(max_dimensions * sample) + dim] << std::endl << std::endl;

      }
   }
   
   hipMemcpy(random_numbers_d, random_numbers, max_dimensions * max_samples * sizeof(double), hipMemcpyHostToDevice);
   compute_distances<<<1, 1>>>(max_samples, max_dimensions, random_numbers_d, distances_d);
   hipDeviceSynchronize();
   hipMemcpy(distances, distances_d, max_samples*sizeof(double), hipMemcpyDeviceToHost);

   for (size_t i = 0; i < 1; ++i) {
      // for (size_t j = 0; j < max_dimensions; ++j) {
         // std::cout << random_numbers[(max_dimensions * i) + j] << std::endl;
      // }
      std::cout << distances[i] << std::endl;
      std::cout << sqrt(distances[i]) << std::endl;
   }

   hipFree(distances_d);
   hipFree(random_numbers_d);
   free(random_numbers);
   free(distances);
}


int main() {
   compute(2, 16, 10'000);
   return 0;
}
//==================================================================== 80 ====>>